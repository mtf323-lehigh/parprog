
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <sys/time.h>
#include <omp.h>

#define N 200000000


__global__
void saxpy(int n, float a, float *x, float *y)
{
  int i = blockIdx.x*blockDim.x + threadIdx.x;
  if (i < n) y[i] = a*x[i] + y[i];
}

int main(void)
{
  float *x, *y, *d_x, *d_y;
  struct timeval start_time, stop_time, elapsed_time;

  x = (float*)malloc(N*sizeof(float));
  y = (float*)malloc(N*sizeof(float));

  hipMalloc(&d_x, N*sizeof(float)); 
  hipMalloc(&d_y, N*sizeof(float));

  for (int i = 0; i < N; i++) {
    x[i] = 1.0f;
    y[i] = 2.0f;
  }

  gettimeofday(&start_time,NULL);
  hipMemcpy(d_x, x, N*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_y, y, N*sizeof(float), hipMemcpyHostToDevice);

  // Perform SAXPY on 1M elements
  saxpy<<<(N+255)/256, 256>>>(N, 2.0f, d_x, d_y);

  hipMemcpy(y, d_y, N*sizeof(float), hipMemcpyDeviceToHost);
  gettimeofday(&stop_time,NULL);
  timersub(&stop_time, &start_time, &elapsed_time);

  printf("Total time was %f seconds.\n", elapsed_time.tv_sec+elapsed_time.tv_usec/1000000.0);

  hipFree(d_x);
  hipFree(d_y);
  free(x);
  free(y);
}
